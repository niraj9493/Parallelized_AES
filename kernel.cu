#include "hip/hip_runtime.h"
//
//  CasAES_CUDA.c
//  CasAES_CUDA
//  Created by Carter McCardwell on 11/11/14.
// Modified by Niraj Surati Nov/5/2018

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <string.h>

#include <hip/hip_runtime.h>

const int Nb_h = 4;
const int Nr_h = 14;
const int Nk_h = 8;

const uint8_t s_h[256] = {
        0x63, 0x7C, 0x77, 0x7B, 0xF2, 0x6B, 0x6F, 0xC5, 0x30, 0x01, 0x67, 0x2B, 0xFE, 0xD7, 0xAB, 0x76,
        0xCA, 0x82, 0xC9, 0x7D, 0xFA, 0x59, 0x47, 0xF0, 0xAD, 0xD4, 0xA2, 0xAF, 0x9C, 0xA4, 0x72, 0xC0,
        0xB7, 0xFD, 0x93, 0x26, 0x36, 0x3F, 0xF7, 0xCC, 0x34, 0xA5, 0xE5, 0xF1, 0x71, 0xD8, 0x31, 0x15,
        0x04, 0xC7, 0x23, 0xC3, 0x18, 0x96, 0x05, 0x9A, 0x07, 0x12, 0x80, 0xE2, 0xEB, 0x27, 0xB2, 0x75,
        0x09, 0x83, 0x2C, 0x1A, 0x1B, 0x6E, 0x5A, 0xA0, 0x52, 0x3B, 0xD6, 0xB3, 0x29, 0xE3, 0x2F, 0x84,
        0x53, 0xD1, 0x00, 0xED, 0x20, 0xFC, 0xB1, 0x5B, 0x6A, 0xCB, 0xBE, 0x39, 0x4A, 0x4C, 0x58, 0xCF,
        0xD0, 0xEF, 0xAA, 0xFB, 0x43, 0x4D, 0x33, 0x85, 0x45, 0xF9, 0x02, 0x7F, 0x50, 0x3C, 0x9F, 0xA8,
        0x51, 0xA3, 0x40, 0x8F, 0x92, 0x9D, 0x38, 0xF5, 0xBC, 0xB6, 0xDA, 0x21, 0x10, 0xFF, 0xF3, 0xD2,
        0xCD, 0x0C, 0x13, 0xEC, 0x5F, 0x97, 0x44, 0x17, 0xC4, 0xA7, 0x7E, 0x3D, 0x64, 0x5D, 0x19, 0x73,
        0x60, 0x81, 0x4F, 0xDC, 0x22, 0x2A, 0x90, 0x88, 0x46, 0xEE, 0xB8, 0x14, 0xDE, 0x5E, 0x0B, 0xDB,
        0xE0, 0x32, 0x3A, 0x0A, 0x49, 0x06, 0x24, 0x5C, 0xC2, 0xD3, 0xAC, 0x62, 0x91, 0x95, 0xE4, 0x79,
        0xE7, 0xC8, 0x37, 0x6D, 0x8D, 0xD5, 0x4E, 0xA9, 0x6C, 0x56, 0xF4, 0xEA, 0x65, 0x7A, 0xAE, 0x08,
        0xBA, 0x78, 0x25, 0x2E, 0x1C, 0xA6, 0xB4, 0xC6, 0xE8, 0xDD, 0x74, 0x1F, 0x4B, 0xBD, 0x8B, 0x8A,
        0x70, 0x3E, 0xB5, 0x66, 0x48, 0x03, 0xF6, 0x0E, 0x61, 0x35, 0x57, 0xB9, 0x86, 0xC1, 0x1D, 0x9E,
        0xE1, 0xF8, 0x98, 0x11, 0x69, 0xD9, 0x8E, 0x94, 0x9B, 0x1E, 0x87, 0xE9, 0xCE, 0x55, 0x28, 0xDF,
        0x8C, 0xA1, 0x89, 0x0D, 0xBF, 0xE6, 0x42, 0x68, 0x41, 0x99, 0x2D, 0x0F, 0xB0, 0x54, 0xBB, 0x16
};

uint8_t Rcon_h[256] = {
        0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a,
        0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39,
        0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a,
        0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8,
        0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef,
        0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc,
        0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b,
        0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3,
        0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94,
        0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04, 0x08, 0x10, 0x20,
        0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63, 0xc6, 0x97, 0x35,
        0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd, 0x61, 0xc2, 0x9f,
        0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d, 0x01, 0x02, 0x04,
        0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36, 0x6c, 0xd8, 0xab, 0x4d, 0x9a, 0x2f, 0x5e, 0xbc, 0x63,
        0xc6, 0x97, 0x35, 0x6a, 0xd4, 0xb3, 0x7d, 0xfa, 0xef, 0xc5, 0x91, 0x39, 0x72, 0xe4, 0xd3, 0xbd,
        0x61, 0xc2, 0x9f, 0x25, 0x4a, 0x94, 0x33, 0x66, 0xcc, 0x83, 0x1d, 0x3a, 0x74, 0xe8, 0xcb, 0x8d
};

__constant__ uint8_t s[256];
__constant__ int Nb;
__constant__ int Nr;
__constant__ int Nk;
__constant__ uint32_t ek[60];

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void cudaDevAssist(hipError_t code, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "cudaDevAssistant: %s %d\n", hipGetErrorString(code), line);
        if (abort) exit(code);
    }
}

uint32_t sw(uint32_t word)
{
    union {
        uint32_t word;
        uint8_t bytes[4];
    }subWord;
    subWord.word = word;

    subWord.bytes[3] = s_h[subWord.bytes[3]];
    subWord.bytes[2] = s_h[subWord.bytes[2]];
    subWord.bytes[1] = s_h[subWord.bytes[1]];
    subWord.bytes[0] = s_h[subWord.bytes[0]];

    return subWord.word;
}

__device__ void sb(uint8_t* in)
{
    for (int i = 0; i < 32; i++) { in[i] = s[in[i]]; }
}

__device__ void sb_st(uint8_t* in)
{
    for (int i = 0; i < 16; i++) { in[i] = s[in[i]]; }

}

__device__ void mc(uint8_t* arr)
{
    for (int i = 0; i < 4; i++)
    {
        uint8_t a[4];
        uint8_t b[4];
        uint8_t c;
        uint8_t h;
        for (c = 0; c < 4; c++) {
            a[c] = arr[(4 * c + i)];
            h = (uint8_t)((signed char)arr[(4 * c + i)] >> 7);
            b[c] = arr[(4 * c + i)] << 1;
            b[c] ^= 0x1B & h;
        }
        arr[(i)] = b[0] ^ a[3] ^ a[2] ^ b[1] ^ a[1];
        arr[(4 + i)] = b[1] ^ a[0] ^ a[3] ^ b[2] ^ a[2];
        arr[(8 + i)] = b[2] ^ a[1] ^ a[0] ^ b[3] ^ a[3];
        arr[(12 + i)] = b[3] ^ a[2] ^ a[1] ^ b[0] ^ a[0];
    }

}

__device__ void sr(uint8_t* arr)
{
    uint8_t out[16];
    //On per-row basis (+1 shift ea row)
    //Row 1
    out[0] = arr[0];
    out[1] = arr[1];
    out[2] = arr[2];
    out[3] = arr[3];
    //Row 2
    out[4] = arr[5];
    out[5] = arr[6];
    out[6] = arr[7];
    out[7] = arr[4];
    //Row 3
    out[8] = arr[10];
    out[9] = arr[11];
    out[10] = arr[8];
    out[11] = arr[9];
    //Row 4
    out[12] = arr[15];
    out[13] = arr[12];
    out[14] = arr[13];
    out[15] = arr[14];

    for (int i = 0; i < 16; i++)
    {
        arr[i] = out[i];
    }
}

uint32_t rw(uint32_t word)
{
    union {
        uint8_t bytes[4];
        uint32_t word;
    } subWord;
    subWord.word = word;

    uint8_t B0 = subWord.bytes[3], B1 = subWord.bytes[2], B2 = subWord.bytes[1], B3 = subWord.bytes[0];
    subWord.bytes[3] = B1; //0
    subWord.bytes[2] = B2; //1
    subWord.bytes[1] = B3; //2
    subWord.bytes[0] = B0; //3

    return subWord.word;
}

void K_Exp(uint8_t* pk, uint32_t* out)
{
    int i = 0;
    union {
        uint8_t bytes[4];
        uint32_t word;
    } temp;
    union {
        uint8_t bytes[4];
        uint32_t word;
    } univar[60];

    for (i = 0; i < Nk_h; i++)
    {
        univar[i].bytes[3] = pk[i * 4];
        univar[i].bytes[2] = pk[i * 4 + 1];
        univar[i].bytes[1] = pk[i * 4 + 2];
        univar[i].bytes[0] = pk[i * 4 + 3];
    }

    for (i = Nk_h; i < Nb_h*(Nr_h + 1); i++)
    {
        temp.word = univar[i - 1].word;
        if (i % Nk_h == 0)
        {
            temp.word = (sw(rw(temp.word)));
            temp.bytes[3] = temp.bytes[3] ^ (Rcon_h[i / Nk_h]);
        }
        else if (Nk_h > 6 && i % Nk_h == 4)
        {
            temp.word = sw(temp.word);
        }
        if (i - 4 % Nk_h == 0)
        {
            temp.word = sw(temp.word);
        }
        univar[i].word = univar[i - Nk_h].word ^ temp.word;
    }
    for (i = 0; i < 60; i++)
    {
        out[i] = univar[i].word;
    }
}

__device__ void ark(uint8_t* state, int strD, uint32_t* eK)
{
    union {
        uint32_t word;
        uint8_t bytes[4];
    } kb[4];

    kb[0].word = eK[strD];
    kb[1].word = eK[strD + 1];
    kb[2].word = eK[strD + 2];
    kb[3].word = eK[strD + 3];

    for (int i = 0; i < 4; i++)
    {
        state[i] = state[i] ^ kb[i].bytes[3];
        state[i + 4] = state[i + 4] ^ kb[i].bytes[2];
        state[i + 8] = state[i + 8] ^ kb[i].bytes[1];
        state[i + 12] = state[i + 12] ^ kb[i].bytes[0];
    }
}

__global__ void cudaRunner(uint8_t *in)
{
    uint8_t state[16];
    int localid = blockDim.x * blockIdx.x + threadIdx.x; //Data is shifted by 16 * ID of worker
    for (int i = 0; i < 16; i++) { state[i] = in[(localid * 16) + i]; }

    ark(state, 0, ek);
    for (int i = 1; i < 14; i++)
    {
        sb_st(state);
        sr(state);
        mc(state);
        ark(state, i*Nb, ek);
    }

    sb_st(state);
    sr(state);
    ark(state, Nr*Nb, ek);

    for (int i = 0; i < 16; i++) { in[(localid * 16) + i] = state[i]; }
}

int main()
{   
    printf("Testing AES-256 -Parallel\n");
    clock_t c_start, c_stop;

    const int RUNNING_THREADS = 0x400;
    const int DATA_PRO_BYTES = (RUNNING_THREADS * 16);
    const float B_TO_MB_DENO = 1024.0 * 1024;
    const long BUFF_SIZE = 0x4E200;
    uint8_t in[BUFF_SIZE];
    const int NUM_FILES=6;
    char *files[] = {
                   "one_MB.txt",
                   "five_MB.txt",
                   "eight_MB.txt",
                   "sixty_four_MB.txt",
                   "hundred_MB.txt",
                   "Two_GB.txt"
    };


    uint8_t key[32] = {  0x60, 0x3d, 0xeb, 0x10, 0x15, 0xca, 0x71, 0xbe, 0x2b, 0x73, 0xae, 0xf0, 0x85, 0x7d, 0x77, 0x81,
                         0x1f, 0x35, 0x2c, 0x07, 0x3b, 0x61, 0x08, 0xd7, 0x2d, 0x98, 0x10, 0xa3, 0x09, 0x14, 0xdf, 0xf4 };
    uint32_t ek_h[60];

     K_Exp(key, ek_h);


    //send constants to GPU
    hipSetDevice(0);
    cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nk), &Nk_h, sizeof(int), 0, hipMemcpyHostToDevice), 535, true);
    cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nr), &Nr_h, sizeof(int), 0, hipMemcpyHostToDevice), 543, true);
    cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(Nb), &Nb_h, sizeof(int), 0, hipMemcpyHostToDevice), 903, true);
    cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(s), &s_h, 256 * sizeof(uint8_t), 0, hipMemcpyHostToDevice), 920, true);
    cudaDevAssist(hipMemcpyToSymbol(HIP_SYMBOL(ek), &ek_h, 60 * sizeof(uint32_t), 0, hipMemcpyHostToDevice), 823, true);
    hipDeviceSynchronize();

    uint8_t *devState = NULL;
    cudaDevAssist(hipMalloc((void**)&devState, RUNNING_THREADS * 16 * sizeof(uint8_t)), 425, true);
    for (int x = 0; x < NUM_FILES; x++)
    {
        FILE* fd = fopen(files[x], "r");
        if (fd == NULL)
        {
            printf("\n Error opening file %s\n", files[x]);
            exit(-1);
        }
        c_start = clock();

        uint8_t states[RUNNING_THREADS][16] = { 0x00 };
        int ch = 0;
        int spawn = 0;
        while (fread(in, sizeof(char), DATA_PRO_BYTES, fd))
        {
            uint8_t* itr = in;
            spawn = 0;
            for (int i = 0; i < RUNNING_THREADS; i++) //Dispatch many control threads that will report back to main (for now 5x) - 1 worker per state
            {
                spawn++;
                memcpy(states[i], itr, 16);
                itr += 16;
            }
            //arrange data correctly
            for (int i = 0; i < spawn; i++)
            {
                uint8_t temp[16];
                memcpy(&temp[0], &states[i][0], sizeof(uint8_t));
                memcpy(&temp[4], &states[i][1], sizeof(uint8_t));
                memcpy(&temp[8], &states[i][2], sizeof(uint8_t));
                memcpy(&temp[12], &states[i][3], sizeof(uint8_t));
                memcpy(&temp[1], &states[i][4], sizeof(uint8_t));
                memcpy(&temp[5], &states[i][5], sizeof(uint8_t));
                memcpy(&temp[9], &states[i][6], sizeof(uint8_t));
                memcpy(&temp[13], &states[i][7], sizeof(uint8_t));
                memcpy(&temp[2], &states[i][8], sizeof(uint8_t));
                memcpy(&temp[6], &states[i][9], sizeof(uint8_t));
                memcpy(&temp[10], &states[i][10], sizeof(uint8_t));
                memcpy(&temp[14], &states[i][11], sizeof(uint8_t));
                memcpy(&temp[3], &states[i][12], sizeof(uint8_t));
                memcpy(&temp[7], &states[i][13], sizeof(uint8_t));
                memcpy(&temp[11], &states[i][14], sizeof(uint8_t));
                memcpy(&temp[15], &states[i][15], sizeof(uint8_t));
                for (int c = 0; c < 16; c++) { memcpy(&states[i][c], &temp[c], sizeof(uint8_t)); }
            }

            //printf("\nCycle!: Spawn = %i", spawn);

            cudaDevAssist(hipMemcpy(devState, *states, spawn * 16 * sizeof(uint8_t), hipMemcpyHostToDevice), 426, true);
            cudaDevAssist(hipDeviceSynchronize(), 268, true);
            cudaRunner <<<8, spawn/32 >>> (devState);

            cudaDevAssist(hipDeviceSynchronize(), 270, true);
            cudaDevAssist(hipMemcpy(*states, devState, spawn * 16 * sizeof(uint8_t), hipMemcpyDeviceToHost), 431, true);

            //printf("%.02f MB\b\b\b\b\b\b\b\b\b", (float)((DATA_PRO_BYTES* ++ch) / B_TO_MB_DENO));
        }//end of while
        fclose(fd);
        c_stop = clock();
        float diff = (((float)c_stop - (float)c_start) / CLOCKS_PER_SEC);

        printf("Time taken for Encrypting %-18s: %.2fs\n",files[x],diff);
    }
    hipFree(devState);
    hipDeviceReset();
    return 0;
}